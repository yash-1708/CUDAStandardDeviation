#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <ctime>
#include <math.h>

#define M 2048//threads per block

__global__ void blockSummer(int *mainVec,int *blockSum,int N,int mean,bool mode){
	__shared__ int sumVec[M+1];
	int tid = threadIdx.x;
	int eid = blockIdx.x * blockDim.x + threadIdx.x;
	if (eid < N) {
		if (mode == true) {
			sumVec[tid] = (mainVec[eid] - mean) * (mainVec[eid] - mean);
		}
		else {
			sumVec[tid] = mainVec[eid];
		}
	}
	else {
		sumVec[tid] = 0;
	}
	__syncthreads();
	for (unsigned int i = 1; i < blockDim.x; i *= 2) {
		if (tid % (2 * i) == 0) {
			sumVec[tid] += sumVec[tid + i];
		}
		__syncthreads();
	}
	if (tid == 0) {
		blockSum[blockIdx.x] = sumVec[0];
	}
}

void serialStdDev(int* mainVec, int* intermediate, int N) {
	int sum = 0;
	for (size_t i = 0; i < N; i++) {
		sum += mainVec[i];
	}

	int mean = sum / N;

	for (size_t i = 0; i < N; i++) {
		intermediate[i] = (mainVec[i] - mean) * (mainVec[i] - mean);
	}

	int sum2 = 0;
	
	for (size_t i = 0; i < N; i++) {
		sum2 += intermediate[i];
	}

	float mean2 = sum2 / N;
	float stddev = sqrt(mean2);
	//printf("\nStandard Deviation : %f",stddev);
}

void randomInts(int* vector, int length) {
	for (size_t i = 0; i < length; i++){
		vector[i] = rand() % 100;
	}
}

int main(){
	clock_t start, stop;
	int N = 95000000;
	int numOfBlocks = (N + M - 1) / M;
	int *mainVec = (int*)malloc(N * sizeof(int));
	int* intermediate = (int*)malloc(N * sizeof(int));
	int *blockSum = (int*)malloc(numOfBlocks * sizeof(int));
	int *blockSum2 = (int*)malloc(numOfBlocks * sizeof(int));
	randomInts(mainVec, N);
	int *d_mainVec, * d_blockSum, * d_blockSum2;
	hipMalloc(&d_mainVec, N * sizeof(int));
	hipMalloc(&d_blockSum, numOfBlocks * sizeof(int));

	//printf("\n*************PARALLEL EXECUTION*************/n");

	hipMemcpy(d_mainVec, mainVec, N * sizeof(int), hipMemcpyHostToDevice);
	start = std::clock();
	blockSummer <<<numOfBlocks, M >>> (d_mainVec, d_blockSum, N, 0, false);
	hipDeviceSynchronize();
	hipMemcpy(blockSum, d_blockSum, numOfBlocks * sizeof(int), hipMemcpyDeviceToHost);
	int sum = 0;
	for (size_t i = 0; i < numOfBlocks; i++){
		sum += blockSum[i];
	}
	int mean = (int)(sum / N);
	blockSummer <<<numOfBlocks, M >>> (d_mainVec, d_blockSum, N, mean, true);
	hipDeviceSynchronize();
	hipMemcpy(blockSum, d_blockSum, numOfBlocks * sizeof(int), hipMemcpyDeviceToHost);
	int stddevsum = 0;
	for (size_t i = 0; i < numOfBlocks; i++) {
		stddevsum += blockSum[i];
	}
	float stddev = (stddevsum / N);
	float finalstddev = sqrt(stddev);
	//printf("\nStandard Deviation : %f", finalstddev);
	stop = std::clock();
	long float timeP = stop - start;
	//printf("\n*************SERIAL EXECUTION*************/n");
	start = std::clock();
	serialStdDev(mainVec, intermediate, N);
	stop = std::clock();
	long float timeN = stop - start;

	//getting GPU properties and storing in prop
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	int cores = prop.multiProcessorCount * 128;
	float totalCost = cores * timeP;

	//results printing
	printf("\n********************************************************************************************************\n");
	printf("N \t\t\t Nor Time \t Par Time \t Cores \t\t Tot Cost \t Speedup \t Efficiency \n");
	printf("%-20d \t %-7.3f \t %-7.3f \t %-10d \t %-7.3f \t %-7.3f \t %-5.5f \n", N, timeN, timeP, cores, totalCost, timeN / timeP, timeN / (timeP * cores));
	printf("\n********************************************************************************************************\n");

	free(mainVec);
	free(blockSum);
	free(intermediate);
	hipFree(d_mainVec);
	hipFree(d_blockSum);

	return 0;
}
